#include <gtest/gtest.h>
#include "../src/ntt.cuh"


TEST(DeviceTests, MaxGridSizeProp0is2147483647) {
    CudaDevice();
    int device = 0; // Assuming you are using device 0
    hipSetDevice(device);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    int expectedResult = 2147483647;
    ASSERT_EQ(prop.maxGridSize[0], expectedResult);

    //std::cout << "Maximum Grid Size: " << prop.maxGridSize[0] << " x " << prop.maxGridSize[1] << " x " << prop.maxGridSize[2] << std::endl;

}

#ifdef BARRETT_64 //ajaveed todo: get rid of ifdef logic altogether

TEST(GpuModularBarret, modOneIsZero) {
    using namespace barrett64_gpu;
    
    Data a = 0, b = 0;
    Modulus c(1);
    BarrettOperations barro;
    
    ASSERT_EQ( barro.add(a, b, c) ,0);
    
}

TEST(GpuModularBarret, modFactorIsZero) {
    /**
     >>> ((1<<16) + (1<<16)) % (1<<16)
     0
    */
    using namespace barrett64_gpu;
    
    Data a = 1 << 16, b = 1 << 16;
    Modulus c(1 << 16);
    BarrettOperations barro;
    std::cout <<"barro.add(a, b, c) " << barro.add(a, b, c) << std::endl;
    ASSERT_EQ( barro.add(a, b, c) ,0);
    
}

#endif

int main(int argc, char** argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}