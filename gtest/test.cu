#include <gtest/gtest.h>
#include "../src/ntt.cuh"


TEST(DeviceTests, MaxGridSizeProp0is2147483647) {
    CudaDevice();
    int device = 0; // Assuming you are using device 0
    hipSetDevice(device);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    int expectedResult = 2147483647;
    ASSERT_EQ(prop.maxGridSize[0], expectedResult);

    //std::cout << "Maximum Grid Size: " << prop.maxGridSize[0] << " x " << prop.maxGridSize[1] << " x " << prop.maxGridSize[2] << std::endl;

}
int main(int argc, char** argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}